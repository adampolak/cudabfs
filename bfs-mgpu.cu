#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
using namespace mgpu;

#include <queue>
#include <vector>
#include <fstream>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <cstdio>
using namespace std;

__global__ void UpdateDistanceAndVisitedKernel(
    const int* __restrict__ frontier, int frontier_size, int d,
    int* distance, int* visited) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < frontier_size; i += step) {
    distance[frontier[i]] = d;
    atomicOr(visited + (frontier[i] >> 5), 1 << (frontier[i] & 31));
  }
}

__global__ void CalculateFrontierStartsAndDegreesKernel(
    const int* __restrict__ nodes, const int* __restrict__ frontier, int n,
    int* node_frontier_starts, int* node_frontier_degrees) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < n; i += step) {
    node_frontier_starts[i] = nodes[frontier[i]];
    node_frontier_degrees[i] = nodes[frontier[i] + 1] - nodes[frontier[i]];
  }
}

__global__ void AdvanceFrontierPhase1Kernel(
      const int* __restrict__ edge_frontier, int edge_frontier_size,
      const int* __restrict__ visited,
      int* parent, int* edge_frontier_success) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < edge_frontier_size; i += step) {
    int v = edge_frontier[i];
    int success = (((visited[v >> 5] >> (v & 31)) & 1) == 0 && parent[v] == -1) ? 1 : 0;
    if (success)
      parent[edge_frontier[i]] = i;
    edge_frontier_success[i] = success;
  }
}

__global__ void AdvanceFrontierPhase2Kernel(
      const int* __restrict__ edge_frontier, int edge_frontier_size,
      const int* __restrict__ parent, int* edge_frontier_success) {
  int from = blockDim.x * blockIdx.x + threadIdx.x;
  int step = gridDim.x * blockDim.x;
  for (int i = from; i < edge_frontier_size; i += step)
    if (edge_frontier_success[i] && parent[edge_frontier[i]] != i)
      edge_frontier_success[i] = 0;
}

void ParallelBFS(
    int n, int m, MGPU_MEM(int) nodes, MGPU_MEM(int) edges, int source,
    MGPU_MEM(int) distance, CudaContext& context) {
  MGPU_MEM(int) visited = context.Fill((n + 31) / 32, 0);
  MGPU_MEM(int) parent = context.Fill(n, -1);
  MGPU_MEM(int) node_frontier = context.Malloc<int>(n);
  MGPU_MEM(int) node_frontier_starts = context.Malloc<int>(n);  
  MGPU_MEM(int) node_frontier_degrees = context.Malloc<int>(n);
  MGPU_MEM(int) edge_frontier = context.Malloc<int>(m);
  MGPU_MEM(int) edge_frontier_success = context.Malloc<int>(m);
  node_frontier->FromHost(&source, 1);
  int node_frontier_size = 1;
  for (int d = 0; node_frontier_size > 0; ++d) {
    // cerr << "d = " << d << " frontier_size = " << node_frontier_size << endl;
    // PrintArray(*node_frontier, "%d", 10);
    UpdateDistanceAndVisitedKernel<<<128, 128, 0, context.Stream()>>>(
        node_frontier->get(), node_frontier_size, d,
        distance->get(), visited->get());
    CalculateFrontierStartsAndDegreesKernel<<<128, 128, 0, context.Stream()>>>(
        nodes->get(), node_frontier->get(), node_frontier_size,
        node_frontier_starts->get(), node_frontier_degrees->get());
    int edge_frontier_size;
    ScanExc(
        node_frontier_degrees->get(), node_frontier_size,
        &edge_frontier_size, context);
    IntervalGather(
        edge_frontier_size, node_frontier_starts->get(),
        node_frontier_degrees->get(), node_frontier_size, edges->get(),
        edge_frontier->get(), context);
    AdvanceFrontierPhase1Kernel<<<128, 128, 0, context.Stream()>>>(
        edge_frontier->get(), edge_frontier_size, visited->get(),
        parent->get(), edge_frontier_success->get());
    AdvanceFrontierPhase2Kernel<<<128, 128, 0, context.Stream()>>>(
        edge_frontier->get(), edge_frontier_size,
        parent->get(), edge_frontier_success->get());
    ScanExc(
        edge_frontier_success->get(), edge_frontier_size,
        &node_frontier_size, context);
    IntervalExpand(
        node_frontier_size, edge_frontier_success->get(),
        edge_frontier->get(), edge_frontier_size,
        node_frontier->get(), context);
  }
}

typedef unsigned long long uint64_t;

uint64_t CalculateChecksum(const vector<int>& distance) {
  uint64_t checksum = 0;
  for (int i = 0; i < distance.size(); ++i)
    if (distance[i] != -1)
      checksum += (uint64_t)i * (uint64_t)distance[i];
  return checksum;
}

uint64_t Time() {
  timespec tp;
  clock_gettime(CLOCK_MONOTONIC_RAW, &tp);
  return (tp.tv_nsec + (uint64_t)1000000000 * tp.tv_sec) / 1000000;
}

uint64_t ParallelBFS(
    const vector<int>& nodes, const vector<int>& edges, int source) {
  ContextPtr context = CreateCudaDevice(0);
  MGPU_MEM(int) dev_nodes = context->Malloc(nodes);
  MGPU_MEM(int) dev_edges = context->Malloc(edges);
  MGPU_MEM(int) dev_distance = context->Fill(nodes.size() - 1, -1);
  uint64_t t = Time();
  ParallelBFS(
      nodes.size() - 1, edges.size(), dev_nodes, dev_edges, source,
      dev_distance, *context);
  t = Time() - t;
  cerr << "GPU: " << t << " ms" << endl;
  vector<int> distance;
  dev_distance->ToHost(distance, nodes.size() - 1);
  return CalculateChecksum(distance);
}

uint64_t SequentialBFS(
    const vector<int>& nodes, const vector<int>& edges, int source) {
  vector<int> distance(nodes.size() - 1, -1);
  uint64_t t = Time();
  distance[source] = 0;
  queue<int> q;
  q.push(source);
  while (!q.empty()) {
    int u = q.front();
    q.pop();
    for (int i = nodes[u]; i < nodes[u + 1]; ++i) {
      int v = edges[i];
      if (distance[v] == -1) {
        distance[v] = distance[u] + 1;
        q.push(v);
      }
    }
  }
  t = Time() - t;
  cerr << "CPU: " << t << " ms" << endl;
  return CalculateChecksum(distance);
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    cerr << "Usage: " << argv[0] << " GRAPH" << endl;
    exit(1);
  }

  ifstream in(argv[1], ios::binary);  
  assert(in.is_open());
  int n, m;
  in.read((char*)&n, sizeof(int));
  in.read((char*)&m, sizeof(int));
  vector<int> nodes(n + 1), edges(m);
  in.read((char*)nodes.data(), nodes.size() * sizeof(int));
  in.read((char*)edges.data(), edges.size() * sizeof(int));

  for (int i = 0; i < 5; ++i) {
    int source = rand() % n;
    uint64_t seqsum = SequentialBFS(nodes, edges, source);
    uint64_t parsum = ParallelBFS(nodes, edges, source);
    assert(seqsum == parsum);
  }
}
